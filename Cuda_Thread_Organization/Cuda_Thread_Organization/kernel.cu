#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void helloWorld()
{
	const int a = blockIdx.x;
	const int b = blockIdx.y;
	const int c = blockIdx.z;
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;
	const int tz = threadIdx.z;
	int tid = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	int bid = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
	int total_tid = bid * blockDim.x * blockDim.y * blockDim.z + tid;
	printf("Hello World from block-(%d, %d, %d)and thread-(%d, %d, %d)!--%d\n", a, b, c, tx, ty, tz, total_tid);
}
int main()
{
	const dim3 block_size(1, 2, 3);
	const dim3 grid_size(2, 4, 2);
	helloWorld << <grid_size, block_size >> > ();
	hipDeviceSynchronize();
}